#include <hip/hip_runtime.h>
#include <stdio.h>


/*
params
a: vector A
b: vector B
c: vector resultado
*/
__global__ void vectorAddGPU(int* a, int* b, int* c){
    //calcular el id del bloque
    int id= blockIdx.x ;
    //sumar cada posicion del vector a y b
    c[id]= a[id] + b[id];
}

int main(){

    //N elementos
    int N = 10;

    //variables para los vectores del host (CPU)
    int *hostA, *hostB, *hostC;

    //variables para los vectores del device (GPU)
    int *deviceA, *deviceB, *deviceC;

    //bytes para los elementos de cada vector
    size_t bytes = sizeof(int) * N;

    //reserva de la memoria para cada vector del Host
    hostA= (int*) malloc(bytes);
    hostB= (int*) malloc(bytes);
    hostC= (int*) malloc(bytes);

    //reserva de la memoria para cada vector del Device
    hipMalloc(&deviceA, bytes);
    hipMalloc(&deviceB, bytes);
    hipMalloc(&deviceC, bytes);

    //inicializacion de los vectores  A y B del host
    for(int i=0; i<N; i++){
        hostA[i] = 3;
        hostB[i] = 1;
    }

    //copia de la memoria de los vectores del host 
    //hacia los vectores del device
    hipMemcpy(deviceA, hostA, bytes, hipMemcpyHostToDevice);
    hipMemcpy(deviceB, hostB, bytes, hipMemcpyHostToDevice);

    //definicion de los bloque e hilos para el kernel
    int nThreads = 1;
    int nBlocks = 10; 

    //llamada al metodo e inicializacion del kernel
    vectorAddGPU<<<nBlocks, nThreads>>>(deviceA, deviceB, deviceC);

    //copia de la memoria del vector resultado del device
    //hacia el vector del resiltado del host
    hipMemcpy(hostC, deviceC, bytes, hipMemcpyDeviceToHost);

    //impresion de resultados
    for(int i=0; i<N; i++){
       printf("%d\t",hostC[i]);
    }

    //liberacion de la memoria de los vectores del device
    hipFree(deviceA);
    hipFree(deviceB);
    hipFree(deviceC);

    return 0;
}