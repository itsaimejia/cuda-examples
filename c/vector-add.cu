#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#include <math.h>

void vectorAddCPU(int* a, int* b, int* c, int n){
    for(int i=0; i<n; i++){
        c[i]= a[i] + b[i];
    }
}

__global__ void vectorAddGPU(int* a, int* b, int* c, int N){
    int id= blockIdx.x * blockDim.x+threadIdx.x;
    if(id<N){
        c[id]= a[id] + b[id];
    }
}

int main(){

    
    int N = 10000;

    int *hostA, *hostB, *hostC;

    int *deviceA, *deviceB, *deviceC;

    size_t bytes = sizeof(int) * N;

    hostA= (int*) malloc(bytes);
    hostB= (int*) malloc(bytes);
    hostC= (int*) malloc(bytes);

    hipMalloc(&deviceA, bytes);
    hipMalloc(&deviceB, bytes);
    hipMalloc(&deviceC, bytes);

    for(int i=0; i<n; i++){
        hostA[i] = i;
        hostB[i] = i;
    }

    hipMemcpy(deviceA, hostA, bytes, hipMemcpyHostToDevice);
    hipMemcpy(deviceB, hostB, bytes, hipMemcpyHostToDevice);

    int nThreads = 128;
    int nBlocks = (int)ceil(N / nThreads); 

    vectorAddGPU<<<nBlocks, nThreads>>>(deviceA, deviceB, deviceC, N);

    hipMemcpy(hostC, deviceC, bytes, hipMemcpyDeviceToHost);

    for(int i=0; i<N; i++){
       printf("%d\t",&c[i]);
    }
    hipFree(deviceA);
    hipFree(deviceB);
    hipFree(deviceC);

    return 0;
}